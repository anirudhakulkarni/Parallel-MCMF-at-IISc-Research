
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
using namespace std;
#include <chrono>
using namespace std::chrono;
void random_ints(int *a, int N)
{
    int i;
    for (i = 0; i < N; ++i)
        a[i] = rand();
}
__global__ void add(int *a, int *b, int *c)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    c[index] = a[index] + b[index];
}

#define N (2048 * 2048 * 100)
#define THREADS_PER_BLOCK 512
int main(void)
{

    auto start = high_resolution_clock::now();
    int *a, *b, *c;       // host copies of a, b, c
    int *d_a, *d_b, *d_c; // device copies of a, b, c
    int size = N * sizeof(int);
    // Alloc space for device copies of a, b, c
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    
    hipMalloc((void **)&d_c, size);
    // Alloc space for host copies of a, b, c and setup input values
    a = (int *)malloc(size);
    b = (int *)malloc(size);
    c = (int *)malloc(size);
    random_ints(b, N);
    random_ints(a, N);
    // Copy inputs to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
    // Launch add() kernel on GPU
    add<<<N / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_a, d_b, d_c);
    // Copy result back to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
    // Cleanup
    free(a);
    free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>(stop - start);
    cout << "Time taken by function: " << duration.count() << " microseconds" << endl;

    return 0;
}